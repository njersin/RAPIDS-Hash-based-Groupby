#include "hip/hip_runtime.h"
#include <stdint.h>
#include <chrono>
#include "hashtable.h"
#include "hashfunction.h"
#include "hashkernel.h"

__constant__ uint32_t c_crc_x64_32_tab[HASH_TAB_SIZE];


template <typename T>
__host__ int getnumdistinctkeys(T* h_key_columns, int num_key_columns, int num_key_rows)
{
    return num_key_rows; //for now return number of rows
}


template <typename T>
__device__ int comparekeyrows(T* d_key_columns, int num_key_columns, int num_key_rows,
                     int a, int b)
{
    int equal = 1;
    for (int i = 0; i < num_key_columns; i++) {
        if (d_key_columns[i * num_key_rows + a] != d_key_columns[i * num_key_rows + b])
            return 0;
    }
    return equal;
}


template <typename T>
__global__ void groupbykernel(T* d_key_columns, int num_key_columns, int num_key_rows,
                              T* d_value_columns, int num_value_columns, int num_value_rows,
                              reduction_op reduct_ops[], int num_ops,
                              hashbucket<T>* d_hashtable, int hash_table_rows, int* d_num_unique_keys)
{
  int row = threadIdx.x + blockIdx.x * blockDim.x;

  if (row < num_key_rows) {

      int bucket_index = crc_x64_32_hash<T>(d_key_columns, num_key_columns, num_key_rows, row) % hash_table_rows;
      int old_key_row, current_key_row;

      int tryagain = 1;
      while (tryagain) {

        current_key_row = d_hashtable[bucket_index].key_row;
        if (current_key_row == EMPTYMARKER) {

          old_key_row = atomicCAS(&d_hashtable[bucket_index].key_row, current_key_row, row);

          if (old_key_row != current_key_row) {
            current_key_row = old_key_row;
          } else {
            tryagain = 0; //key was inserted, proceed to update reduction fields
            current_key_row = row;
            atomicAdd(d_num_unique_keys, 1); //update count of unique keys
          }
        }

        if (current_key_row != row) {
          //compare rows
          if (comparekeyrows<T>(d_key_columns, num_key_rows, num_key_columns, current_key_row, row)) {
            tryagain = 0; //found matching bucket, proceed to update reduction fields
          } else {
            tryagain = 1; //collision, try next bucket
            bucket_index = (bucket_index + 1) % hash_table_rows;
          }
        }
        __syncthreads();
      }

      //update reduction fields
      for (int i = 0; i < num_ops; i++) {
        if (reduct_ops[i] == max) {
          atomicMax(&d_hashtable[bucket_index].max, d_value_columns[i * num_value_rows + row]);
        } else if (reduct_ops[i] == min) {
          atomicMin(&d_hashtable[bucket_index].min, d_value_columns[i * num_value_rows + row]);
        } else if (reduct_ops[i] == sum) {
          atomicAdd(&d_hashtable[bucket_index].sum, d_value_columns[i * num_value_rows + row]);
        } else if (reduct_ops[i] == count) {
          atomicAdd(&d_hashtable[bucket_index].count, 1);
        }
      }

  }
}


template <typename T>
__global__ void getouputdatakernel(T* d_output_keys, int num_key_columns, int num_key_rows,
                                   T* d_output_values, int num_value_columns, int num_value_rows,
                                   hashbucket<T>* d_hashtable, int num_unique_keys, int hash_table_rows,
                                   reduction_op reduct_ops[], int num_ops, T* d_key_columns)
{
    int output_row = threadIdx.x + blockIdx.x * blockDim.x;

    if (output_row < num_unique_keys) {

        int scan_size = (hash_table_rows / num_unique_keys);
        int num_scan_rows = scan_size;
        if ((output_row = num_unique_keys - 1) && (num_key_rows % num_unique_keys)) {
            num_scan_rows += 1;
        }

        int start_row = output_row * scan_size;
        hashbucket<T> bucket;

        for (int i = 0; i < num_scan_rows; i++) {
            start_row += i;
            bucket = d_hashtable[start_row];
            if (bucket.key_row != EMPTYMARKER) {

                //copy row
                for (int j = 0; j < num_key_columns; j++) {
                    d_output_keys[j * num_unique_keys + output_row] = d_key_columns[j * num_key_rows + bucket.key_row];
                }

                //copy reduction values
                for (int k = 0; k < num_ops; k++) {
                    if (reduct_ops[k] == max) {
                        d_output_values[k * num_unique_keys + output_row] = bucket.max;
                    } else if (reduct_ops[k] == min) {
                        d_output_values[k * num_unique_keys + output_row] = bucket.min;
                    } else if (reduct_ops[k] == sum) {
                        d_output_values[k * num_unique_keys + output_row] = bucket.sum;
                    } else if (reduct_ops[k] == count) {
                        d_output_values[k * num_unique_keys + output_row] = bucket.count;
                    }
                }
            }
        }
    }
}


template <typename T>
__host__ struct output_data groupby(T* h_key_columns, int num_key_columns, int num_key_rows,
                                    T* h_value_columns, int num_value_columns, int num_value_rows,
                                    reduction_op ops[], int num_ops)
{

  //get number of unique keys
  int* h_num_unique_keys, d_num_unique_keys;
  hipMalloc((void **) &d_num_unique_keys, sizeof(int));
  int hash_table_rows = getnumdistinctkeys<T>(h_key_columns, num_key_columns, num_key_rows);

  //allocate memory for hash table on device
  hashbucket<T>* d_hashtable;
  hashtablesize = hash_table_rows * sizeof(hashbucket<T>);
  hipMalloc((void **) &d_hashtable, hashtablesize);

  //initialize hash table
  init_hash_table<T>(d_hashtable, hash_table_rows);

  //transfer keys and values data to device
  T* d_key_columns, d_value_columns;
  int num_key_pitch, num_value_pitch;

  int key_data_size = num_key_rows * num_key_columns * sizeof(T);
  hipMalloc((void **)&d_key_columns, key_data_size);
  hipMemcpy(d_key_columns, h_key_columns, key_data_size, hipMemcpyHostToDevice);

  int value_data_size = num_value_rows * num_value_columns * sizeof(T);
  hipMalloc((void **)&d_value_columns, value_data_size);
  hipMemcpy(d_value_columns, h_value_columns, value_data_size, hipMemcpyHostToDevice);

  //copy hash key tab to constant memory
  cudaMemcpytoSymbol(c_crc_x64_32_tab, crc_x64_32_tab, HASH_TAB_SIZE * sizeof(uint32_t));

  //launch reduction kernel
  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  dim3 dimGrid((num_key_rows + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);
  groupbykernel<T><<<dimGrid, dimBlock>>>(d_key_columns, num_key_columns, num_key_rows,
                                          d_value_columns, num_value_columns, num_value_rows,
                                          reduct_ops, num_ops,
                                          d_hashtable, hash_table_rows, d_num_unique_keys);
  hipDeviceSynchronize();

  //copy number of unique keys from device memory
  hipMemcpy(h_num_unique_keys, d_num_unique_keys, sizeof(int), hipMemcpyDeviceToHost);

  //allocate space on host memory for output keys and output values
  int output_key_size = h_num_unique_keys * num_key_columns * sizeof(T);
  int output_values_size = h_num_unique_keys * num_value_columns * sizeof(T);

  T* h_output_keys, h_output_values;
  hipHostMalloc(&h_output_keys, output_key_size);
  hipHostMalloc(&h_output_values, output_values_size);

  T* d_output_keys, d_output_values;
  hipMalloc((void **) &d_output_keys, output_key_size);
  hipMalloc((void **) &d_output_values, output_values_size);

  //launch kernel to summarize results in output format
  dim3 dimGrid((h_num_unique_keys + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);
  getouputdatakernel<T><<<dimGrid, dimBlock>>>(d_output_keys, num_key_columns, num_key_rows,
                                               d_output_values, num_value_columns, num_value_rows
                                               d_hashtable, h_num_unique_keys, hash_table_rows,
                                               reduct_ops, num_ops, d_key_columns);

  //copy results back to host
  hipMemcpy(h_output_keys, d_output_keys, output_key_size, hipMemcpyDeviceToHost);
  hipMemcpy(h_output_values, d_output_values, output_values_size, hipMemcpyDeviceToHost);

  struct output_data<T> output;
  output.keys = h_output_keys;
  output.values = h_output_values;
  output.unique_keys = h_num_unique_keys;

  //free device memory
  hipFree(d_num_unique_keys);
  hipFree(d_hashtable);
  hipFree(d_key_columns);
  hipFree(d_value_columns);
  hipFree(d_output_keys);
  hipFree(d_output_values);

  return output;
}
